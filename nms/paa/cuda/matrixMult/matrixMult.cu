#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <fstream> // load from file
#include <iostream> // std::cout
#include <sstream> // load from file
#include <string> // load from file
#include <time.h> // evals comparison
#include <vector> // matrix values storing
#include <algorithm> //std::max
#include <numeric> //std::accumulate for vector sum

#include <hip/hip_runtime.h>
#include <>


// matrix dimension setting for the whole program
// shared memory in CUDA opt. requires compile time const. - global const.
// also define type of values in matrix for further optimization
// GPUs are optimized for single precision floating point - float is better than int or double
const int max_dim = 1 << 10;
typedef float matrix_vals_type;


// simple matrix structure using human friendly 2d vectors for values
struct simpleMatrix {
	unsigned int n = 0;
	unsigned int m = 0;
	std::vector<std::vector<matrix_vals_type>> values;

	simpleMatrix() {};
	simpleMatrix(unsigned int m,unsigned int n)
		:m(m), n(n)
	{
		values.resize(m, std::vector<matrix_vals_type>(n));
	}
};

// optimized matrix structure using 1d vectors for values
struct optMatrix {
	int n = 0;
	std::vector<matrix_vals_type> values; // optimization by using 1d vector

	optMatrix() {};
	optMatrix(const int n)
		:n(n)
	{
		values.resize(n * n);
	}
};


// load simple 2d matrix into a vector from a file
simpleMatrix loadSimpleMatrix(std::string filename) {
	std::ifstream input_file(filename);
	if (input_file.is_open()) {
		simpleMatrix matrix {};
		std::string line;

		while (std::getline(input_file, line)) {
			matrix_vals_type value;
			std::vector<matrix_vals_type> row;
			std::istringstream line_stream(line);
			while (line_stream >> value) {
				row.push_back(value);
			}
			matrix.values.push_back(row);
		}

		matrix.m = matrix.values.size();
		matrix.n = matrix.values[0].size();
		return matrix;
	}
	else {
		printf("Matrix could not be loaded! No such file: %s", filename);
	}
}


// generate random simple matrix of size m, n with int values from 1 to max_val
std::vector<std::vector<matrix_vals_type>> getRandomSimpleMatrixValues(const int& m, const int& n, const int& max_val) {
	std::vector<std::vector<matrix_vals_type>> values(m, std::vector<matrix_vals_type>(n));
	for (int i = 0; i < m; i++) {
		for (int j = 0; j < n; j++) {
			values[i][j] = rand() % max_val + 1;
		}
	}
	return values;
}


// overload to have same simpleMatrix values as already obtained optMatrix values
std::vector<std::vector<matrix_vals_type>> getRandomSimpleMatrixValues(const int& m, const int& n, const optMatrix& matrix) {
	std::vector<std::vector<matrix_vals_type>> values(m, std::vector<matrix_vals_type>(n));
	for (int i = 0; i < m; i++) {
		for (int j = 0; j < n; j++) {
			values[i][j] = matrix.values[i * max_dim + j];
		}
	}
	return values;
}

// generate random opt. matrix of size n, n with int values from 1 to max_val
std::vector<matrix_vals_type> getRandomOptMatrixValues(const int& n, const int& max_val) {
	std::vector<matrix_vals_type> values(n * n);
	std::generate(values.begin(), values.end(), [max_val]() {return rand() % max_val + 1; });
	return values;
}


// print simple matrix to console
void printSimpleMatrix(simpleMatrix& matrix) {
	for (int i = 0; i < matrix.m; i++) {
		for (int j = 0; j < matrix.n; j++) {
			std::cout << matrix.values[i][j] << " ";
		}
		std::cout << std::endl;
	}
	std::cout << std::endl;
}

// print opt. matrix to console
void printOptMatrix(optMatrix& matrix) {
	for (int i = 0; i < matrix.n * matrix.n; i += max_dim) {
		for (int j = 0; j < matrix.n; j++) {
			std::cout << matrix.values[i + j] << " ";
		}
		std::cout << std::endl;
	}
	std::cout << std::endl;
}


// check whether simple matrices have compatible dimensions
inline bool checkDims(simpleMatrix& matrix_A, simpleMatrix& matrix_B) {
	return (matrix_A.n == matrix_B.m);
}


// multiply matrices using standard CPU multiplication
// most naive approach using 2d vectors (still faster than 2d dynamic arrays though)
// still passing by reference though
// unsigned is slower than int
simpleMatrix matrixSimpleMultCPU(simpleMatrix& matrix_A, simpleMatrix& matrix_B) {
	simpleMatrix matrix_C(matrix_A.m, matrix_B.n);

	for (unsigned int i = 0; i < matrix_A.m; i++) {
		for (unsigned int j = 0; j < matrix_B.n; j++) {
			for (unsigned int k = 0; k < matrix_A.n; k++) {
				matrix_C.values[i][j] += matrix_A.values[i][k] * matrix_B.values[k][j];
			}
		}
	}

	return matrix_C;
}


// optimized cpu matrix multiplication
// 1d vectors with optimized indexing instead of 2d vectors
// using int instead of unsigned is faster
// not accesing optMatrix struct in for cycles by substituting int n = matrix_C.n
// declaring iterators ahead of cycles, k = k_block is faster than k = 0
// using submatrix_C instead of accesing matrix_C structure
// switching k (cols of A, rows of B) and j (rows of A, cols of B) cycles - switching j most frequently is better for cache access as c++ is row major language
// using block submatrices for faster cache access
// using pragma unroll
optMatrix matrixOptMultCPU(optMatrix& matrix_A, optMatrix& matrix_B) {
	optMatrix matrix_C(matrix_A.n);
	int n = matrix_C.n;

	int submatrix_size = 1 << 8;
	int k_block = 0, j_block = 0, i = 0, k = k_block, j = j_block;

	matrix_vals_type submatrix_A = 0;
	for (k_block = 0; k_block < n; k_block += submatrix_size) {
		for (j_block = 0; j_block < n; j_block += submatrix_size) {
			for (i = 0; i < n; i++) {
				for (k = k_block; k < k_block + submatrix_size; k++) {
					// A[i][k]
					submatrix_A = matrix_A.values[i * n + k];
#pragma unroll(8)
					for (j = j_block; j < j_block + submatrix_size; j++) {
						// C[i][j] = A[i][k] * B[k][j]
						matrix_C.values[i * n + j] += submatrix_A * matrix_B.values[k * n + j];
					}
				}
			}
		}
	}

	/*int i = 0, int j = 0, int k = 0;
	matrix_vals_type submatrix_C = 0;
	for (i = 0; i < n; i++) {
		for (j = 0; j < n; j++) {
			submatrix_C = 0;
#pragma unroll
			for (k = 0; k < n; k++) {
				submatrix_C += matrix_A.values[i * n + k] * matrix_B.values[k * n + j];
			}
			matrix_C.values[i * n + j] = submatrix_C;
		}
	}*/

	/*int i = 0, int j = 0, int k = 0;
	matrix_vals_type submatrix_C = 0;
	for (i = 0; i < n; i++) {
	for (k = 0; k < n; k++) {
		//submatrix_C = 0;
		for (j = 0; j < n; j++) {
			matrix_C.values[i * n + j] += matrix_A.values[i * n + k] * matrix_B.values[k * n + j];
			//submatrix_C += matrix_A.values[i * n + k] * matrix_B.values[k * n + j];
		}
		//matrix_C.values[i * n + j] = submatrix_C;
	}
	}*/

	return matrix_C;
}

// simple CUDA matrix multiplication
// indexing by CUDA blocks and threads equivalent to CPU opt. multiplication
// CUDA kernel cannot return anything - void, memcpy in driver necessary
// kernel is introduced using __global__ keyword
__global__ 
void matrixSimpleMultCUDA(const int max_size, const matrix_vals_type* matrix_a, const matrix_vals_type* matrix_b, matrix_vals_type* matrix_c) {
	// row and col indices from <blocks, threads> kernel input
	// blockDim = #of threads in a block
	// blockIdx = number of block within its specific grid (represents matrix here)
	// threadIdx = number of thread within its specific block
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	matrix_vals_type submatrix_c = 0;
	if (row < max_size && col < max_size) {
		// we access rows and cols through blocks and threads and have to iterate through both to get one element of resulting matrix
		for (int i = 0; i < max_size; i++) {
			// [row][i], [i][col]
			submatrix_c += matrix_a[row * max_size + i] * matrix_b[i * max_size + col];
		}
	}
	matrix_c[row * max_size + col] = submatrix_c;
}


// optimized CUDA matrix multiplication
// using matrix_vals_type float - faster than double or int
// using 1d static shared memory submatrices 
// unrolling multiplication for cycle
// using vars for threadIdx, blockIdx, blockDim improves performance slightly - not worth it?
// (i + tiy) * max_size is better than version withou parantheses
__global__
void matrixOptMultCUDA(const int max_size, const matrix_vals_type* matrix_a, const matrix_vals_type* matrix_b, matrix_vals_type* matrix_c) {
	// declaring these vars in advance provides slight performance improvement
	int tiy = threadIdx.y;
	int tix = threadIdx.x;
	int biy = blockIdx.y;
	int bix = blockIdx.x;
	int bdy = blockDim.y;
	int bdx = blockDim.x;

	int row = biy * bdy + tiy;
	int col = bix * bdy + tix;

	// using (static) shared memory is faster than global memory
	// https://developer.nvidia.com/blog/using-shared-memory-cuda-cc/
	// all threads in the block have access to the same shared memory
	// To ensure correct results when parallel threads cooperate, we must synchronize the threads. CUDA provides a simple barrier synchronization primitive, __syncthreads(). A thread�s execution can only proceed past a __syncthreads() after all threads in its block have executed the __syncthreads()
	// could also be implemented as 2d - no improvement
	__shared__ matrix_vals_type shared_a[max_dim];
	__shared__ matrix_vals_type shared_b[max_dim];

	matrix_vals_type submatrix_c = 0;
	int i = 0, j = 0;
	for (i = 0; i < max_size; i += bdx) {
		// shared memory is relevant for threads in one block ergo indexing as [tiy][tix] -> [tiy * bdx + tix]
		// further indexing by i subdivisions [row][i + tix], [i + tiy][col]
		shared_a[tiy * bdy + tix] = matrix_a[row * max_size + i + tix];
		shared_b[tiy * bdy + tix] = matrix_b[col + (i + tiy) * max_size];

		// synchronize before computation
		__syncthreads();

		// try unrolling cycle
		// j is defacto threadIdx bounded by its blockDim
#pragma unroll
		for (j = 0; j < bdx; j++) {
			// [tiy][j], [j][tix]
			submatrix_c += shared_a[tiy * bdy + j] * shared_b[j * bdx + tix];
		}

		// synchronize after computation
		__syncthreads();
	}

	matrix_c[row * max_size + col] = submatrix_c;
}

// CUDA matrix multiplication preparation and driver
optMatrix matrixMultCUDADriver(bool opt, const int num_threads, const int num_blocks, size_t byte_size, optMatrix& matrix_A, optMatrix& matrix_B) {
	optMatrix matrix_C(matrix_A.n);

	// alloc. memory in CUDA capable nVidia GPU
	matrix_vals_type* matrix_a, * matrix_b, * matrix_c;
	hipMalloc(&matrix_a, byte_size);
	hipMalloc(&matrix_b, byte_size);
	hipMalloc(&matrix_c, byte_size);

	// copy data from CPU (host) to GPU (device)
	hipMemcpy(matrix_a, matrix_A.values.data(), byte_size, hipMemcpyHostToDevice);
	hipMemcpy(matrix_b, matrix_B.values.data(), byte_size, hipMemcpyHostToDevice);

	// creating 2d grids for matrix rows and cols
	dim3 threads(num_threads, num_threads);
	dim3 blocks(num_blocks, num_blocks);

	// using optimized or unonptimized variant of CUDA multiplication kernel
	if(!opt)
		matrixSimpleMultCUDA << <blocks, threads >> > (max_dim, matrix_a, matrix_b, matrix_c);
	else
		matrixOptMultCUDA << <blocks, threads>> > (max_dim, matrix_a, matrix_b, matrix_c);
	
	// copy result back from GPU to matrix_C on CPU
	hipMemcpy(matrix_C.values.data(), matrix_c, byte_size, hipMemcpyDeviceToHost);

	// free GPU memory
	hipFree(matrix_a);
	hipFree(matrix_b);
	hipFree(matrix_c);

	return matrix_C;
}

// check whether the results are the same
inline bool checkResults(optMatrix& matrix_1, optMatrix& matrix_2) {
	//return (matrix_1.values == matrix_2.values);

	int errs = 0;
	for (int i = 0; i < matrix_1.values.size(); i++) {
		matrix_1.values[i] = std::round(matrix_1.values[i]);
		matrix_2.values[i] = std::round(matrix_2.values[i]);
		if (matrix_1.values[i] != matrix_2.values[i]) {
			printf("Warning: value mismatch: %f vs. %f on index: %i \n", matrix_1.values[i], matrix_2.values[i], i);
			errs++;
		}
		
	}
	if (errs > 0)
		return false;
	else
		return true;
}


// calculate multiplication elapsed time
inline double calcElapsedTime(const clock_t& start_time, const clock_t& end_time) {
	return ((double)(end_time - start_time)) / CLOCKS_PER_SEC;
}


// calculate multiplication average time
inline double calcAvgTime(const int num_evals, std::vector<double> measured_times) {
	return std::accumulate(measured_times.begin(), measured_times.end(), decltype(measured_times)::value_type(0)) / num_evals;
}


// calculate optimization time saving
inline double calcSaving(const double& opt_avg_time, const double& simple_avg_time) {
	return (1 - opt_avg_time / simple_avg_time) * 100;
}

// calculate optimization performance improvement
inline double calcPerf(const double& opt_avg_time, const double& simple_avg_time) {
	return 1 / (opt_avg_time / simple_avg_time);
}

int main() {
	// number of evaluation runs
	int num_evals = 10;

	// load matrices from files
	//matrix_A = loadMatrix("matrix_A.txt");
    //matrix_B = loadMatrix("matrix_B.txt");

	// generate random matrices of following specs
	int max_value = 100;
	int matrix_A_m = max_dim;
	int matrix_A_n = max_dim;
	int matrix_B_m = matrix_A_n;
	int matrix_B_n = max_dim;
	//int max_dim = std::max(std::max(matrix_A_m, matrix_A_n), matrix_B_n);

	// CUDA parameters
	size_t byte_size = max_dim * max_dim * sizeof(matrix_vals_type);
	int num_blocks = 32;
	int num_threads = max_dim / num_blocks;

	// initialize matrices
	simpleMatrix simple_matrix_A(matrix_A_m, matrix_A_n);
	simpleMatrix simple_matrix_B(matrix_B_m, matrix_B_n);
	simpleMatrix simple_matrix_C(matrix_A_m, matrix_B_n);

	optMatrix opt_matrix_A(matrix_A_n);
	optMatrix opt_matrix_B(matrix_A_n);
	optMatrix opt_matrix_C_cpu(matrix_A_n);
	optMatrix opt_matrix_C_cuda_simple(matrix_A_n);
	optMatrix opt_matrix_C_cuda_opt(matrix_A_n);

	// generate random matrices
	std::srand(time(NULL));
	/*simple_matrix_A.values = getRandomSimpleMatrixValues(matrix_A_m, matrix_A_n, max_value);
	simple_matrix_B.values = getRandomSimpleMatrixValues(matrix_B_m, matrix_B_n, max_value);
	opt_matrix_A.values = getRandomOptMatrixValues(matrix_A_n, max_value);
	opt_matrix_B.values = getRandomOptMatrixValues(matrix_B_n, max_value);*/

	printf("CPU, CUDA matrix multiplication of max. dim: %d comparison: \n", max_dim);
	if (!checkDims(simple_matrix_A, simple_matrix_B)) {
		printf("Matrices have incompatible size for multiplication! \n Exiting...");
		return 1;
	}
	else {
		// debug print
		//printf("Multiplying matrices: \n");
		//printf("Matrix A: \n");
		//printMatrix(matrix_A);
		//printf("Matrix B: \n");
		//printMatrix(matrix_B);

		// initialize vectors for time measuring
		std::vector<double> simple_cpu_times_arr(num_evals, 0);
		std::vector<double> simple_cpu_times_vect(num_evals, 0);
		std::vector<double> opt_cpu_times(num_evals, 0);
		std::vector<double> simple_cuda_times(num_evals, 0);
		std::vector<double> opt_cuda_times(num_evals, 0);

		// initialize time variables
		clock_t start_time, end_time;
		double elapsed_time;

		for (int i = 0; i < num_evals; i++) {
			opt_matrix_A.values = getRandomOptMatrixValues(matrix_A_n, max_value);
			opt_matrix_B.values = getRandomOptMatrixValues(matrix_B_n, max_value);
			simple_matrix_A.values = getRandomSimpleMatrixValues(matrix_A_m, matrix_A_n, opt_matrix_A);
			simple_matrix_B.values = getRandomSimpleMatrixValues(matrix_B_m, matrix_B_n, opt_matrix_B);



			printf("Cycle %d/%d: \n", i + 1, num_evals);

			// simple CPU multiplication evals
			start_time = clock();
			simple_matrix_C = matrixSimpleMultCPU(simple_matrix_A, simple_matrix_B);
			end_time = clock();
			elapsed_time = calcElapsedTime(start_time, end_time);
			simple_cpu_times_vect.push_back(elapsed_time);
			printf("simple CPU vector multiplication time [s]: %4.3f \n", elapsed_time);
			//printf("Result, matrix C: \n");
			//printSimpleMatrix(simple_matrix_C);
			
			// opt. CPU multiplication evals
			start_time = clock();
			opt_matrix_C_cpu = matrixOptMultCPU(opt_matrix_A, opt_matrix_B);
			end_time = clock();
			elapsed_time = calcElapsedTime(start_time, end_time);
			opt_cpu_times.push_back(elapsed_time);
			printf("opt. CPU multiplication time [s]: %4.3f \n", elapsed_time);
			//printf("Result, matrix C: \n");
			//printOptMatrix(opt_matrix_C);

			// simple CUDA multiplication evals
			bool opt = false;
			start_time = clock();
			opt_matrix_C_cuda_simple = matrixMultCUDADriver(opt, num_threads, num_blocks, byte_size, opt_matrix_A, opt_matrix_B);
			end_time = clock();
			elapsed_time = calcElapsedTime(start_time, end_time);
			simple_cuda_times.push_back(elapsed_time);
			printf("simple CUDA multiplication time [s]: %4.3f \n", elapsed_time);
			//printf("Result, matrix C: \n");
			//printOptMatrix(opt_matrix_C);
			if (!checkResults(opt_matrix_C_cpu, opt_matrix_C_cuda_simple))
				printf("Warning: Results differ! \n");

			// opt. CUDA multiplication evals
			opt = true;
			start_time = clock();
			opt_matrix_C_cuda_opt = matrixMultCUDADriver(opt, num_threads, num_blocks, byte_size, opt_matrix_A, opt_matrix_B);
			end_time = clock();
			elapsed_time = calcElapsedTime(start_time, end_time);
			opt_cuda_times.push_back(elapsed_time);
			printf("opt. CUDA multiplication time [s]: %4.3f \n \n", elapsed_time);
			//printf("Result, matrix C: \n");
			//printOptMatrix(opt_matrix_C);
			if (!checkResults(opt_matrix_C_cpu, opt_matrix_C_cuda_opt))
				printf("Warning: Results differ! \n");
		}

		// summary of runs
		printf("\nSummary: \n");

		double simple_cpu_avg_time = calcAvgTime(num_evals, simple_cpu_times_vect);
		printf("Avg. simple CPU multiplication time [s] of matrices of max. dim.: %d over %d iters.: %4.3f \n", max_dim, num_evals, simple_cpu_avg_time);
		
		double opt_cpu_avg_time = calcAvgTime(num_evals, opt_cpu_times);
		printf("Avg. opt. CPU multiplication time [s] of matrices of max. dim.: %d over %d iters.: %4.3f \n", max_dim, num_evals, opt_cpu_avg_time);

		double simple_cuda_avg_time = calcAvgTime(num_evals, simple_cuda_times);
		printf("Avg. simple CUDA multiplication time [s] of matrice of max. dim.: %d over %d iters.: %4.3f \n", max_dim, num_evals, simple_cuda_avg_time);

		double opt_cuda_avg_time = calcAvgTime(num_evals, opt_cuda_times);
		printf("Avg. opt. CUDA multiplication time [s] of matrice of max. dim.: %d over %d iters.: %4.3f \n", max_dim, num_evals, opt_cuda_avg_time);

		/*double cpu_opt_saving = calcSaving(opt_cpu_avg_time, simple_cpu_avg_time);
		printf("CPU optimization improves speed by app. %4.2f %%! \n", cpu_opt_saving);

		double cuda_opt_saving = calcSaving(opt_cuda_avg_time, simple_cuda_avg_time);
		printf("CUDA optimization improves speed by app. %4.2f %%! \n", cuda_opt_saving);

		double total_opt_saving = calcSaving(opt_cuda_avg_time, simple_cpu_avg_time);
		printf("Total optimization improves speed by app. %4.2f %%! \n", total_opt_saving);*/

		double cpu_perf_imp = calcPerf(opt_cpu_avg_time, simple_cpu_avg_time);
		printf("CPU optimization improves performace %4.2f times! \n", cpu_perf_imp);

		double cuda_perf_imp = calcPerf(opt_cuda_avg_time, simple_cuda_avg_time);
		printf("CUDA optimization improves performace %4.2f times! \n", cuda_perf_imp);

		double total_perf_imp = calcPerf(opt_cuda_avg_time, simple_cpu_avg_time);
		printf("Total optimization improves performace %4.2f times! \n", total_perf_imp);

		// result debug print
		//printf("Result, matrix C: \n");
		//printOptMatrix(opt_matrix_C);
		return 0;
	}
}